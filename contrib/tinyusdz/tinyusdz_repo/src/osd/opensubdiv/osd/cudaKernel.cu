#include "hip/hip_runtime.h"
//
//   Copyright 2013 Pixar
//
//   Licensed under the Apache License, Version 2.0 (the "Apache License")
//   with the following modification; you may not use this file except in
//   compliance with the Apache License and the following modification to it:
//   Section 6. Trademarks. is deleted and replaced with:
//
//   6. Trademarks. This License does not grant permission to use the trade
//      names, trademarks, service marks, or product names of the Licensor
//      and its affiliates, except as required to comply with Section 4(c) of
//      the License and to reproduce the content of the NOTICE file.
//
//   You may obtain a copy of the Apache License at
//
//       http://www.apache.org/licenses/LICENSE-2.0
//
//   Unless required by applicable law or agreed to in writing, software
//   distributed under the Apache License with the above modification is
//   distributed on an "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
//   KIND, either express or implied. See the Apache License for the specific
//   language governing permissions and limitations under the Apache License.
//

#include <assert.h>
#define OSD_PATCH_BASIS_CUDA
#include "../osd/patchBasisCommonTypes.h"
#include "../osd/patchBasisCommon.h"
#include "../osd/patchBasisCommonEval.h"

// -----------------------------------------------------------------------------
template<int N> struct DeviceVertex {

    float v[N];

    __device__ void addWithWeight(DeviceVertex<N> const & src, float weight) {
#pragma unroll
        for(int i = 0; i < N; ++i){
            v[i] += src.v[i] * weight;
        }
    }

    __device__ void clear() {
#pragma unroll
        for(int i = 0; i < N; ++i){
            v[i] = 0.0f;
        }
    }
};

// Specialize DeviceVertex for N=0 to avoid compile error:
// "flexible array member in otherwise empty struct"
template<> struct DeviceVertex<0> {
    __device__ void addWithWeight(DeviceVertex<0> &src, float weight) {}
    __device__ void clear() {}
};

// -----------------------------------------------------------------------------

__device__ void clear(float *dst, int count)
{
    for(int i = 0; i < count; ++i) dst[i] = 0;
}

__device__ void addWithWeight(float *dst, float const *src, float weight, int count)
{
    for(int i = 0; i < count; ++i) dst[i] += src[i] * weight;
}

// --------------------------------------------------------------------------------------------

template <int NUM_ELEMENTS> __global__ void
computeStencils(float const * cvs, float * vbuffer,
                int const * sizes,
                int const * offsets,
                int const * indices,
                float const * weights,
                int start, int end) {

    DeviceVertex<NUM_ELEMENTS> const * src =
        (DeviceVertex<NUM_ELEMENTS> const *)cvs;

    DeviceVertex<NUM_ELEMENTS> * verts =
        (DeviceVertex<NUM_ELEMENTS> *)vbuffer;

    int first = start + threadIdx.x + blockIdx.x*blockDim.x;

    for (int i=first; i<end; i += blockDim.x * gridDim.x) {

        int const * lindices = indices + offsets[i];
        float const * lweights = weights + offsets[i];

        DeviceVertex<NUM_ELEMENTS> dst;
        dst.clear();

        for (int j=0; j<sizes[i]; ++j) {
            dst.addWithWeight(src[lindices[j]], lweights[j]);
        }
        verts[i] = dst;
    }
}

__global__ void
computeStencils(float const * cvs, float * dst,
                int length,
                int srcStride,
                int dstStride,
                int const * sizes,
                int const * offsets,
                int const * indices,
                float const * weights,
                int start, int end) {

    int first = start + threadIdx.x + blockIdx.x*blockDim.x;

    for (int i=first; i<end; i += blockDim.x * gridDim.x) {

        int const * lindices = indices + offsets[i];
        float const * lweights = weights + offsets[i];

        float * dstVert = dst + i*dstStride;
        clear(dstVert, length);

        for (int j=0; j<sizes[i]; ++j) {

            float const * srcVert = cvs + lindices[j]*srcStride;

            addWithWeight(dstVert, srcVert, lweights[j], length);
        }
    }
}

// -----------------------------------------------------------------------------

#define USE_NVIDIA_OPTIMIZATION
#ifdef USE_NVIDIA_OPTIMIZATION

template< int NUM_ELEMENTS, int NUM_THREADS_PER_BLOCK >
__global__ void computeStencilsNv(float const *__restrict cvs,
                                  float * vbuffer,
                                  int const *__restrict sizes,
                                  int const *__restrict offsets,
                                  int const *__restrict indices,
                                  float const *__restrict weights,
                                  int start,
                                  int end)
{
  // Shared memory to stage indices/weights.
  __shared__ int   smem_indices_buffer[NUM_THREADS_PER_BLOCK];
  __shared__ float smem_weights_buffer[NUM_THREADS_PER_BLOCK];

  // The size of a single warp.
  const int WARP_SIZE = 32;
  // The number of warps per block.
  const int NUM_WARPS_PER_BLOCK = NUM_THREADS_PER_BLOCK / WARP_SIZE;
  // The number of outputs computed by a single warp.
  const int NUM_OUTPUTS_PER_WARP = WARP_SIZE / NUM_ELEMENTS;
  // The number of outputs computed by a block of threads.
  const int NUM_OUTPUTS_PER_BLOCK = NUM_OUTPUTS_PER_WARP*NUM_WARPS_PER_BLOCK;
  // The number of active threads in a warp.
  const int NUM_ACTIVE_THREADS_PER_WARP = NUM_OUTPUTS_PER_WARP * NUM_ELEMENTS;

  // The number of the warp inside the block.
  const int warpId = threadIdx.x / WARP_SIZE;
  const int laneId = threadIdx.x % WARP_SIZE;

  // We use NUM_ELEMENTS threads per output. Find which output/element a thread works on.
  int outputIdx = warpId*NUM_OUTPUTS_PER_WARP + laneId/NUM_ELEMENTS, elementIdx = laneId%NUM_ELEMENTS;

  // Each output corresponds to a section of shared memory.
  volatile int   *smem_indices = &smem_indices_buffer[warpId*WARP_SIZE + (laneId/NUM_ELEMENTS)*NUM_ELEMENTS];
  volatile float *smem_weights = &smem_weights_buffer[warpId*WARP_SIZE + (laneId/NUM_ELEMENTS)*NUM_ELEMENTS];

  // Disable threads that have nothing to do inside the warp.
  int i = end;
  if( laneId < NUM_ACTIVE_THREADS_PER_WARP )
    i = start + blockIdx.x*NUM_OUTPUTS_PER_BLOCK + outputIdx;

  // Iterate over the vertices.
  for( ; i < end ; i += gridDim.x*NUM_OUTPUTS_PER_BLOCK )
  {
    // Each thread computes an element of the final vertex.
    float x = 0.f;

    // Load the offset and the size for each vertex. We have NUM_THREADS_PER_VERTEX threads loading the same value.
    const int offset_i = offsets[i], size_i = sizes[i];

    // Iterate over the stencil.
    for( int j = offset_i, j_end = offset_i+size_i ; j < j_end ; )
    {
      int j_it = j + elementIdx;

      // Load some indices and some weights. The transaction is coalesced.
      smem_indices[elementIdx] = j_it < j_end ? indices[j_it] : 0;
      smem_weights[elementIdx] = j_it < j_end ? weights[j_it] : 0.f;

      // Thread now collaborates to load the vertices.
      #pragma unroll
      for( int k = 0 ; k < NUM_ELEMENTS ; ++k, ++j )
        if( j < j_end )
          x += smem_weights[k] * cvs[smem_indices[k]*NUM_ELEMENTS + elementIdx];
    }

    // Store the vertex.
    vbuffer[NUM_ELEMENTS*i + elementIdx] = x;
  }
}

template< int NUM_THREADS_PER_BLOCK >
__global__ void computeStencilsNv_v4(float const *__restrict cvs,
                                     float * vbuffer,
                                     int const *__restrict sizes,
                                     int const *__restrict offsets,
                                     int const *__restrict indices,
                                     float const *__restrict weights,
                                     int start,
                                     int end)
{
  // Iterate over the vertices.
  for( int i = start + blockIdx.x*NUM_THREADS_PER_BLOCK + threadIdx.x ; i < end ; i += gridDim.x*NUM_THREADS_PER_BLOCK )
  {
    // Each thread computes an element of the final vertex.
    float4 x = make_float4(0.f, 0.f, 0.f, 0.f);

    // Iterate over the stencil.
    for( int j = offsets[i], j_end = offsets[i]+sizes[i] ; j < j_end ; ++j )
    {
      float w = weights[j];
      float4 tmp = reinterpret_cast<const float4 *>(cvs)[indices[j]];
      x.x += w*tmp.x;
      x.y += w*tmp.y;
      x.z += w*tmp.z;
      x.w += w*tmp.w;
    }

    // Store the vertex.
    reinterpret_cast<float4*>(vbuffer)[i] = x;
  }
}

#endif // USE_NVIDIA_OPTIMIZATION

// -----------------------------------------------------------------------------

__global__ void
computePatches(const float *src, float *dst,
               float *dstDu, float *dstDv,
               float *dstDuu, float *dstDuv, float *dstDvv,
               int length, int srcStride, int dstStride,
               int dstDuStride, int dstDvStride,
               int dstDuuStride, int dstDuvStride, int dstDvvStride,
               int numPatchCoords, const OsdPatchCoord *patchCoords,
               const OsdPatchArray *patchArrayBuffer,
               const int *patchIndexBuffer,
               const OsdPatchParam *patchParamBuffer) {

    int first = threadIdx.x + blockIdx.x * blockDim.x;

    // PERFORMANCE: not yet optimized

    for (int i = first; i < numPatchCoords; i += blockDim.x * gridDim.x) {

        OsdPatchCoord const &coord = patchCoords[i];
        int arrayIndex = coord.arrayIndex;
        int patchIndex = coord.patchIndex;

        OsdPatchArray const &array = patchArrayBuffer[arrayIndex];
        OsdPatchParam const &param = patchParamBuffer[patchIndex];

        int patchType = OsdPatchParamIsRegular(param)
                ? array.regDesc : array.desc;

        float wP[20], wDu[20], wDv[20], wDuu[20], wDuv[20], wDvv[20];
        int nPoints = OsdEvaluatePatchBasis(patchType, param,
                coord.s, coord.t, wP, wDu, wDv, wDuu, wDuv, wDvv);

        int indexBase = array.indexBase + array.stride *
                (patchIndex - array.primitiveIdBase);

        const int *cvs = patchIndexBuffer + indexBase;

        float * dstVert = dst + i * dstStride;
        clear(dstVert, length);
        for (int j = 0; j < nPoints; ++j) {
            const float * srcVert = src + cvs[j] * srcStride;
            addWithWeight(dstVert, srcVert, wP[j], length);
        }
        if (dstDu) {
            float *d = dstDu + i * dstDuStride;
            clear(d, length);
            for (int j = 0; j < nPoints; ++j) {
                const float * srcVert = src + cvs[j] * srcStride;
                addWithWeight(d, srcVert, wDu[j], length);
            }
        }
        if (dstDv) {
            float *d = dstDv + i * dstDvStride;
            clear(d, length);
            for (int j = 0; j < nPoints; ++j) {
                const float * srcVert = src + cvs[j] * srcStride;
                addWithWeight(d, srcVert, wDv[j], length);
            }
        }
        if (dstDuu) {
            float *d = dstDuu + i * dstDuuStride;
            clear(d, length);
            for (int j = 0; j < nPoints; ++j) {
                const float * srcVert = src + cvs[j] * srcStride;
                addWithWeight(d, srcVert, wDuu[j], length);
            }
        }
        if (dstDuv) {
            float *d = dstDuv + i * dstDuvStride;
            clear(d, length);
            for (int j = 0; j < nPoints; ++j) {
                const float * srcVert = src + cvs[j] * srcStride;
                addWithWeight(d, srcVert, wDuv[j], length);
            }
        }
        if (dstDvv) {
            float *d = dstDvv + i * dstDvvStride;
            clear(d, length);
            for (int j = 0; j < nPoints; ++j) {
                const float * srcVert = src + cvs[j] * srcStride;
                addWithWeight(d, srcVert, wDvv[j], length);
            }
        }
    }
}

// -----------------------------------------------------------------------------

#include "../version.h"

#define OPT_KERNEL(NUM_ELEMENTS, KERNEL, X, Y, ARG) \
    if (length==NUM_ELEMENTS && srcStride==length && dstStride==length) {   \
        KERNEL<NUM_ELEMENTS><<<X,Y>>>ARG;             \
        return;                                     \
    }

#ifdef USE_NVIDIA_OPTIMIZATION
#define OPT_KERNEL_NVIDIA(NUM_ELEMENTS, KERNEL, X, Y, ARG) \
    if (length==NUM_ELEMENTS && srcStride==length && dstStride==length) {   \
        int gridDim = min(X, (end-start+Y-1)/Y); \
        KERNEL<NUM_ELEMENTS, Y><<<gridDim, Y>>>ARG; \
        return;                                     \
    }
#endif

extern "C" {

void CudaEvalStencils(
    const float *src, float *dst,
    int length, int srcStride, int dstStride,
    const int * sizes, const int * offsets, const int * indices,
    const float * weights,
    int start, int end) {
    if (length == 0 || srcStride == 0 || dstStride == 0 || (end <= start)) {
        return;
    }

#ifdef USE_NVIDIA_OPTIMIZATION
    OPT_KERNEL_NVIDIA(3, computeStencilsNv, 2048, 256,
                      (src, dst, sizes, offsets, indices, weights, start, end));
    //OPT_KERNEL_NVIDIA(4, computeStencilsNv, 2048, 256,
    //                  (cvs, dst, sizes, offsets, indices, weights, start, end));
    if (length == 4 && srcStride == length && dstStride == length) {
      int gridDim = min(2048, (end-start+256-1)/256);
      computeStencilsNv_v4<256><<<gridDim, 256>>>(
          src, dst, sizes, offsets, indices, weights, start, end);
      return;
    }
#else
    OPT_KERNEL(3, computeStencils, 512, 32,
               (src, dst, sizes, offsets, indices, weights, start, end));
    OPT_KERNEL(4, computeStencils, 512, 32,
               (src, dst, sizes, offsets, indices, weights, start, end));
#endif

    // generic case (slow)
    computeStencils <<<512, 32>>>(
        src, dst, length, srcStride, dstStride,
        sizes, offsets, indices, weights, start, end);
}

// -----------------------------------------------------------------------------

void CudaEvalPatches(
    const float *src, float *dst,
    int length, int srcStride, int dstStride,
    int numPatchCoords, const OsdPatchCoord *patchCoords,
    const OsdPatchArray *patchArrayBuffer,
    const int *patchIndexBuffer,
    const OsdPatchParam *patchParamBuffer) {

    // PERFORMANCE: not optimized at all

    computePatches <<<512, 32>>>(
        src, dst, NULL, NULL, NULL, NULL, NULL,
        length, srcStride, dstStride, 0, 0, 0, 0, 0,
        numPatchCoords, patchCoords,
        patchArrayBuffer, patchIndexBuffer, patchParamBuffer);
}

void CudaEvalPatchesWithDerivatives(
    const float *src, float *dst,
    float *dstDu, float *dstDv,
    float *dstDuu, float *dstDuv, float *dstDvv,
    int length, int srcStride, int dstStride,
    int dstDuStride, int dstDvStride,
    int dstDuuStride, int dstDuvStride, int dstDvvStride,
    int numPatchCoords, const OsdPatchCoord *patchCoords,
    const OsdPatchArray *patchArrayBuffer,
    const int *patchIndexBuffer,
    const OsdPatchParam *patchParamBuffer) {

    // PERFORMANCE: not optimized at all

    computePatches <<<512, 32>>>(
        src, dst, dstDu, dstDv, dstDuu, dstDuv, dstDvv,
        length, srcStride, dstStride,
        dstDuStride, dstDvStride, dstDuuStride, dstDuvStride, dstDvvStride,
        numPatchCoords, patchCoords,
        patchArrayBuffer, patchIndexBuffer, patchParamBuffer);
}

}  /* extern "C" */
